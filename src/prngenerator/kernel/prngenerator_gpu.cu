#include "hip/hip_runtime.h"
#include <iostream>

#include "../prngenerator_gpu.hpp"

#include "cuda_common/cuPrintf.cu"

namespace locusta {

    template<typename TFloat>
    __global__ void
    gpu_setup_curand(uint64_t seed,
                     hiprandState *curand_states,
                     uint32_t num_generators)
    {
        uint64_t id =
            blockIdx.y * gridDim.x * blockDim.x +
            blockIdx.x * blockDim.x +
            threadIdx.x;

        while(id < num_generators)
        {
            hiprand_init(id + seed, 0, 0, &curand_states[id]);
            id += blockDim.x * gridDim.x + blockDim.y * blockDim.y;
        }
    }

    template<typename TFloat>
    void gpu_setup_curand_dispatch(uint64_t seed,
                                  hiprandState *curand_states,
                                  uint32_t num_generators)
    {
        uint32_t blocks = 32;
        gpu_setup_curand<TFloat> <<<32, 32>>>
            (seed, curand_states, num_generators);
        CudaCheckError();
    }

    // Explicit specialization
    template void gpu_setup_curand_dispatch<float>(uint64_t seed,
                                                  hiprandState *curand_states,
                                                  uint32_t num_generators);

    template void gpu_setup_curand_dispatch<double>(uint64_t seed,
                                                   hiprandState *curand_states,
                                                   uint32_t num_generators);

}
