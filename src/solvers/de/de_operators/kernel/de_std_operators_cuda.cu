#include "hip/hip_runtime.h"
#include "cuda_common/cuda_helpers.h"
#include "../de_std_operators_cuda_impl.hpp"

namespace locusta {

  /// GPU Kernels Shared Memory Pointer.
  extern __shared__ int de_operators_shared_memory[];

  template <typename TFloat>
  __global__
  void de_whole_crossover_kernel(const uint32_t DIMENSIONS,
                                 const TFloat CROSSOVER_RATE,
                                 const TFloat DIFFERENTIAL_SCALE_FACTOR,
                                 const TFloat * __restrict__ VAR_RANGES,
                                 const TFloat * __restrict__ prn_array,
                                 const uint32_t * __restrict__ trial_selection,
                                 const TFloat * __restrict__ current_vectors,
                                 TFloat * __restrict__ trial_vectors,
                                 hiprandState * __restrict__ local_generator) {

    const uint32_t i = blockIdx.x;
    const uint32_t j = threadIdx.x;

    const uint32_t ISLES = gridDim.x;
    const uint32_t AGENTS = blockDim.x;

    const uint32_t THREAD_OFFSET = ISLES * AGENTS;
    const uint32_t BASE_IDX = i * AGENTS + j;

    hiprandState local_state = local_generator[BASE_IDX];

    const TFloat * agent_prns = prn_array + BASE_IDX;

    const TFloat * target_vector = current_vectors + BASE_IDX;
    TFloat * trial_vector = trial_vectors + BASE_IDX;

    const uint32_t DIFFERENCE_VECTOR_A_IDX = trial_selection[BASE_IDX];
    const uint32_t DIFFERENCE_VECTOR_B_IDX = trial_selection[BASE_IDX + THREAD_OFFSET];
    const uint32_t BASE_VECTOR_IDX = trial_selection[BASE_IDX + (THREAD_OFFSET * 2)];

    const uint32_t DIFFERENCE_A_OFFSET = i * AGENTS + DIFFERENCE_VECTOR_A_IDX;
    const uint32_t DIFFERENCE_B_OFFSET = i * AGENTS + DIFFERENCE_VECTOR_B_IDX;
    const uint32_t BASE_VECTOR_OFFSET =  i * AGENTS + BASE_VECTOR_IDX;

    const TFloat * difference_a_vector = current_vectors + DIFFERENCE_A_OFFSET;
    const TFloat * difference_b_vector = current_vectors + DIFFERENCE_B_OFFSET;
    const TFloat * base_vector = current_vectors + BASE_VECTOR_OFFSET;

    const bool FORCE_PARAMETER_COPY_FLAG = (*agent_prns);
    agent_prns += THREAD_OFFSET; // Advance pointer

    for(uint32_t k = 0; k < DIMENSIONS; ++k) {
      trial_vector[k * THREAD_OFFSET] = difference_a_vector[k * THREAD_OFFSET];
      trial_vector[k * THREAD_OFFSET] -= difference_b_vector[k * THREAD_OFFSET];
      trial_vector[k * THREAD_OFFSET] *= DIFFERENTIAL_SCALE_FACTOR;
      trial_vector[k * THREAD_OFFSET] += base_vector[k * THREAD_OFFSET];
    }

    for(uint32_t k = 0; k < DIMENSIONS; ++k) {
      const bool CROSSOVER_FLAG = (*agent_prns) < CROSSOVER_RATE;
      agent_prns += THREAD_OFFSET; // Advance pointer

      if ((k != FORCE_PARAMETER_COPY_FLAG && !CROSSOVER_FLAG)) {
        trial_vector[k * THREAD_OFFSET] = target_vector[k * THREAD_OFFSET];
      }
    }

    local_generator[BASE_IDX] = local_state;
  }

  template <typename TFloat>
  void de_whole_crossover_dispatch
  (const uint32_t ISLES,
   const uint32_t AGENTS,
   const uint32_t DIMENSIONS,
   const TFloat CROSSOVER_RATE,
   const TFloat DIFFERENTIAL_SCALE_FACTOR,
   const TFloat * VAR_RANGES,
   const TFloat * prn_array,
   const uint32_t * trial_selection,
   const TFloat * current_vectors,
   TFloat * trial_vectors,
   prngenerator_cuda<TFloat> * local_generator) {

    hiprandState * device_generators = local_generator->get_device_generator_states();

    de_whole_crossover_kernel
      <<<ISLES, AGENTS>>>
      (DIMENSIONS,
       CROSSOVER_RATE,
       DIFFERENTIAL_SCALE_FACTOR,
       VAR_RANGES,
       prn_array,
       trial_selection,
       current_vectors,
       trial_vectors,
       device_generators);

    CudaCheckError();
  }

  template
  void de_whole_crossover_dispatch<float>
  (const uint32_t ISLES,
   const uint32_t AGENTS,
   const uint32_t DIMENSIONS,
   const float CROSSOVER_RATE,
   const float DIFFERENTIAL_SCALE_FACTOR,
   const float * VAR_RANGES,
   const float * prn_array,
   const uint32_t * trial_selection,
   const float * current_vectors,
   float * trial_vectors,
   prngenerator_cuda<float> * local_generator);

  template
  void de_whole_crossover_dispatch<double>
  (const uint32_t ISLES,
   const uint32_t AGENTS,
   const uint32_t DIMENSIONS,
   const double CROSSOVER_RATE,
   const double DIFFERENTIAL_SCALE_FACTOR,
   const double * VAR_RANGES,
   const double * prn_array,
   const uint32_t * trial_selection,
   const double * current_vectors,
   double * trial_vectors,
   prngenerator_cuda<double> * local_generator);

  template <typename TFloat>
  __global__
  void de_random_selection_kernel(const TFloat * __restrict__ prn_array,
                                  uint32_t * __restrict__ recombination_idx_array,
                                  uint32_t * __restrict__ recombination_reservoir_array) {

    const uint32_t i = blockIdx.x; // ISLE
    const uint32_t j = threadIdx.x; // AGENT

    const uint32_t ISLES = gridDim.x;
    const uint32_t AGENTS = blockDim.x;

    const uint32_t THREAD_OFFSET = ISLES * AGENTS;
    const uint32_t BASE_IDX = i * AGENTS + j;

    const TFloat * agent_prns = prn_array + BASE_IDX;

    uint32_t * local_candidates = recombination_reservoir_array + BASE_IDX;

    // Resevoir Sampling
    const uint32_t RANDOM_VECTORS = 3;
    const uint32_t SAMPLE_SIZE = RANDOM_VECTORS;

    for(uint32_t k = 0; k < (AGENTS - 1); ++k) {
      if (k < RANDOM_VECTORS) {
        // Fill
        local_candidates[k * THREAD_OFFSET] = k < j ? k : k + 1;
      } else {
        uint32_t r;
        r = (*agent_prns) * (k + 1);
        agent_prns += THREAD_OFFSET; // Advance pointer
        if (r < SAMPLE_SIZE) {
          // Replace
          local_candidates[r * THREAD_OFFSET] = k < j ? k : k + 1;
        }
      }
    }

    for (uint32_t k = 0; k < RANDOM_VECTORS; ++k) {
      recombination_idx_array[BASE_IDX + k * THREAD_OFFSET] = local_candidates[k * THREAD_OFFSET];
    }
  }

  template <typename TFloat>
  void de_random_selection_dispatch
  (const uint32_t ISLES,
   const uint32_t AGENTS,
   const TFloat * prn_array,
   uint32_t * recombination_idx_array,
   uint32_t * recombination_reservoir_array) {
    de_random_selection_kernel
      <<<ISLES, AGENTS>>>
      (prn_array,
       recombination_idx_array,
       recombination_reservoir_array);

    CudaCheckError();
  }

  template
  void de_random_selection_dispatch<float>
  (const uint32_t ISLES,
   const uint32_t AGENTS,
   const float * prn_array,
   uint32_t * recombination_idx_array,
   uint32_t * recombination_reservoir_array);

  template
  void de_random_selection_dispatch<double>
  (const uint32_t ISLES,
   const uint32_t AGENTS,
   const double * prn_array,
   uint32_t * recombination_idx_array,
   uint32_t * recombination_reservoir_array);

}
