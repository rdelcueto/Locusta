#include "hip/hip_runtime.h"
#include "../de_solver_cuda.hpp"
#include "cuda_common/cuda_helpers.h"

namespace locusta {
/// GPU Kernels Shared Memory Pointer.
extern __shared__ int solver_shared_memory[];

/**
 * @brief CUDA kernel for replacing the trial vector.
 *
 * This kernel replaces the trial vector with the best candidate solution.
 *
 * @param DIMENSIONS Number of dimensions per agent.
 * @param previous_vectors Device array of previous vectors.
 * @param previous_fitness Device array of previous fitness values.
 * @param trial_vectors Device array of trial vectors.
 * @param trial_fitness Device array of trial fitness values.
 */
template<typename TFloat>
__global__ void
trial_vector_replace_kernel(const uint32_t DIMENSIONS,
                            TFloat* __restrict__ previous_vectors,
                            const TFloat* __restrict__ previous_fitness,
                            const TFloat* __restrict__ trial_vectors,
                            TFloat* __restrict__ trial_fitness)
{

  const uint32_t i = blockIdx.x;
  const uint32_t j = threadIdx.x;

  const uint32_t ISLES = gridDim.x;
  const uint32_t AGENTS = blockDim.x;

  const uint32_t THREAD_OFFSET = ISLES * AGENTS;
  const uint32_t BASE_IDX = i * AGENTS + j;

  if (trial_fitness[BASE_IDX] > previous_fitness[BASE_IDX]) {
    const TFloat* trial_vector = trial_vectors + BASE_IDX;
    TFloat* target_vector = previous_vectors + BASE_IDX;

    for (uint32_t k = 0; k < DIMENSIONS; ++k) {
      target_vector[k * THREAD_OFFSET] = trial_vector[k * THREAD_OFFSET];
    }
  } else {
    trial_fitness[BASE_IDX] = previous_fitness[BASE_IDX];
  }
}

/**
 * @brief Dispatch function for replacing the trial vector.
 *
 * @param ISLES Number of isles in the population.
 * @param AGENTS Number of agents per isle.
 * @param DIMENSIONS Number of dimensions per agent.
 * @param previous_vectors Array of previous vectors.
 * @param previous_fitness Array of previous fitness values.
 * @param trial_vectors Array of trial vectors.
 * @param trial_fitness Array of trial fitness values.
 */
template<typename TFloat>
void
trial_vector_replace_dispatch(const uint32_t ISLES,
                              const uint32_t AGENTS,
                              const uint32_t DIMENSIONS,
                              TFloat* previous_vectors,
                              const TFloat* previous_fitness,
                              const TFloat* trial_vectors,
                              TFloat* trial_fitness)
{
  trial_vector_replace_kernel<<<ISLES, AGENTS>>>(DIMENSIONS,
                                                 previous_vectors,
                                                 previous_fitness,
                                                 trial_vectors,
                                                 trial_fitness);

  CudaCheckError();
}

template void
trial_vector_replace_dispatch<float>(const uint32_t ISLES,
                                     const uint32_t AGENTS,
                                     const uint32_t DIMENSIONS,
                                     float* previous_vectors,
                                     const float* previous_fitness,
                                     const float* trial_vectors,
                                     float* trial_fitness);

template void
trial_vector_replace_dispatch<double>(const uint32_t ISLES,
                                      const uint32_t AGENTS,
                                      const uint32_t DIMENSIONS,
                                      double* previous_vectors,
                                      const double* previous_fitness,
                                      const double* trial_vectors,
                                      double* trial_fitness);
}
