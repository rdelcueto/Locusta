#include "hip/hip_runtime.h"
#include "cuda_common/cuda_helpers.h"
#include "../ga_std_operators_cuda_impl.hpp"

namespace locusta {

    /// GPU Kernels Shared Memory Pointer.
    extern __shared__ int ga_operators_shared_memory[];

    template <typename TFloat>
    __global__
    void whole_crossover_kernel(const uint32_t DIMENSIONS,
                                const TFloat DEVIATION,
                                const TFloat CROSSOVER_RATE,
                                const TFloat MUTATION_RATE,
                                const uint32_t DIST_LIMIT,
                                const TFloat INV_DIST_LIMIT,
                                const TFloat * __restrict__ VAR_RANGES,
                                const TFloat * __restrict__ prn_array,
                                const uint32_t * __restrict__ couple_selection,
                                const TFloat * __restrict__ parent_genomes,
                                TFloat * __restrict__ offspring_genomes,
                                hiprandState * __restrict__ local_generator) {

        const uint32_t i = blockIdx.x;
        const uint32_t j = threadIdx.x;

        const uint32_t ISLES = gridDim.x;
        const uint32_t AGENTS = blockDim.x;

        const uint32_t THREAD_OFFSET = ISLES * AGENTS;
        const uint32_t BASE_IDX = i * AGENTS + j;

        hiprandState local_state = local_generator[BASE_IDX];

        const TFloat * agent_prns = prn_array + BASE_IDX;

        TFloat * offspring = offspring_genomes + BASE_IDX;
        const TFloat * parentA = parent_genomes + BASE_IDX;

        const bool CROSSOVER_FLAG = (*agent_prns) < CROSSOVER_RATE;
        agent_prns += THREAD_OFFSET; // Advance pointer

        for(uint32_t k = 0; k < DIMENSIONS; ++k) {
            offspring[k * THREAD_OFFSET] = parentA[k * THREAD_OFFSET];
        }

        if (CROSSOVER_FLAG) {
            const uint32_t couple_idx = couple_selection[BASE_IDX];
            const TFloat * parentB = parent_genomes + couple_idx + i * AGENTS;

            for(uint32_t k = 0; k < DIMENSIONS; ++k) {
                offspring[k * THREAD_OFFSET] *= 0.5;
                offspring[k * THREAD_OFFSET] += parentB[k * THREAD_OFFSET] * 0.5;
            }
        }

        for(uint32_t k = 0; k < DIMENSIONS; ++k) {
            const bool GENE_MUTATE_FLAG = (*agent_prns) < MUTATION_RATE;
            agent_prns += THREAD_OFFSET; // Advance pointer

            if(GENE_MUTATE_FLAG) {
                // TODO: MOVE TO SHARED MEM -> BROADCAST
                const TFloat & range = VAR_RANGES[k];

                TFloat x = 0.0;
                for(uint32_t n = 0; n < DIST_LIMIT; ++n) {
                    x += hiprand_uniform(&local_state);
                }

                x *= INV_DIST_LIMIT;
                x -= 0.5;
                x *= DEVIATION * range;

                offspring[k * THREAD_OFFSET] += x;
            }
        }

        local_generator[BASE_IDX] = local_state;
    }

    template <typename TFloat>
    void whole_crossover_dispatch
    (const uint32_t ISLES,
     const uint32_t AGENTS,
     const uint32_t DIMENSIONS,
     const TFloat DEVIATION,
     const TFloat CROSSOVER_RATE,
     const TFloat MUTATION_RATE,
     const uint32_t DIST_LIMIT,
     const TFloat * VAR_RANGES,
     const TFloat * prn_array,
     const uint32_t * couple_selection,
     const TFloat * parent_genomes,
     TFloat * offspring_genomes,
     prngenerator_cuda<TFloat> * local_generator) {

        hiprandState * device_generators = local_generator->get_device_generator_states();
        const TFloat INV_DIST_LIMIT = 1.0 / DIST_LIMIT;

        whole_crossover_kernel
            <<<ISLES, AGENTS>>>
            (DIMENSIONS,
             DEVIATION,
             CROSSOVER_RATE,
             MUTATION_RATE,
             DIST_LIMIT,
             INV_DIST_LIMIT,
             VAR_RANGES,
             prn_array,
             couple_selection,
             parent_genomes,
             offspring_genomes,
             device_generators);

        CudaCheckError();
    }

    template
    void whole_crossover_dispatch<float>
    (const uint32_t ISLES,
     const uint32_t AGENTS,
     const uint32_t DIMENSIONS,
     const float DEVIATION,
     const float CROSSOVER_RATE,
     const float MUTATION_RATE,
     const uint32_t DIST_LIMIT,
     const float * VAR_RANGES,
     const float * prn_array,
     const uint32_t * couple_selection,
     const float * parent_genomes,
     float * offspring_genomes,
     prngenerator_cuda<float> * local_generator);

    template
    void whole_crossover_dispatch<double>
    (const uint32_t ISLES,
     const uint32_t AGENTS,
     const uint32_t DIMENSIONS,
     const double DEVIATION,
     const double CROSSOVER_RATE,
     const double MUTATION_RATE,
     const uint32_t DIST_LIMIT,
     const double * VAR_RANGES,
     const double * prn_array,
     const uint32_t * couple_selection,
     const double * parent_genomes,
     double * offspring_genomes,
     prngenerator_cuda<double> * local_generator);

    template <typename TFloat>
    __global__
    void tournament_selection_kernel(const uint32_t SELECTION_SIZE,
                                     const TFloat SELECTION_P,
                                     const TFloat * __restrict__ fitness_array,
                                     const TFloat * __restrict__ prn_array,
                                     uint32_t * __restrict__ couple_idx_array,
                                     uint32_t * __restrict__ candidates_reservoir_array) {

        const uint32_t i = blockIdx.x; // ISLE
        const uint32_t j = threadIdx.x; // AGENT

        const uint32_t ISLES = gridDim.x;
        const uint32_t AGENTS = blockDim.x;

        const uint32_t THREAD_OFFSET = ISLES * AGENTS;
        const uint32_t BASE_IDX = i * AGENTS + j;

        const TFloat * agent_prns = prn_array + BASE_IDX;

        uint32_t * local_candidates = candidates_reservoir_array + BASE_IDX;

        // Resevoir Sampling
        for(uint32_t k = 0; k < (AGENTS - 1); ++k) {
          if (k < SELECTION_SIZE) {
            // Fill
            local_candidates[k * THREAD_OFFSET] = k < j ? k : k + 1;
          } else {
            uint32_t r;
            r = (*agent_prns) * (k + 1);
            agent_prns += THREAD_OFFSET; // Advance pointer
            if (r < SELECTION_SIZE) {
              // Replace
              local_candidates[r * THREAD_OFFSET] = k < j ? k : k + 1;
            }
          }
        }

        // Tournament
        bool switch_flag;

        uint32_t best_idx = *(local_candidates);
        TFloat best_fitness = fitness_array[best_idx + i * AGENTS];

        // TODO: Check prng cardinality.
        // SELECTION_SIZE - 1

        for(uint32_t k = 1; k < SELECTION_SIZE; ++k) {
            const uint32_t candidate_idx = local_candidates[k * THREAD_OFFSET];
            const TFloat candidate_fitness = fitness_array[candidate_idx + i * AGENTS];

            switch_flag = (candidate_fitness > best_fitness);

            if((SELECTION_P != 0.0f) &&
               (SELECTION_P >= (*agent_prns))) {
                switch_flag = !switch_flag;
            }

            agent_prns += THREAD_OFFSET; // Advance pointer

            if(switch_flag) {
                best_fitness = candidate_fitness;
                best_idx = candidate_idx;
            }
        }

        couple_idx_array[BASE_IDX] = best_idx;
    }


    template <typename TFloat>
    void tournament_selection_dispatch
    (const uint32_t ISLES,
     const uint32_t AGENTS,
     const uint32_t SELECTION_SIZE,
     const TFloat SELECTION_P,
     const TFloat * fitness_array,
     const TFloat * prn_array,
     uint32_t * couple_idx_array,
     uint32_t * candidates_reservoir_array) {
        tournament_selection_kernel
            <<<ISLES, AGENTS>>>
            (SELECTION_SIZE,
             SELECTION_P,
             fitness_array,
             prn_array,
             couple_idx_array,
             candidates_reservoir_array);

        CudaCheckError();
    }

    template
    void tournament_selection_dispatch<float>
    (const uint32_t ISLES,
     const uint32_t AGENTS,
     const uint32_t SELECTION_SIZE,
     const float SELECTION_P,
     const float * fitness_array,
     const float * prn_array,
     uint32_t * couple_idx_array,
     uint32_t * candidates_reservoir_array);

    template
    void tournament_selection_dispatch<double>
    (const uint32_t ISLES,
     const uint32_t AGENTS,
     const uint32_t SELECTION_SIZE,
     const double SELECTION_P,
     const double * fitness_array,
     const double * prn_array,
     uint32_t * couple_idx_array,
     uint32_t * candidates_reservoir_array);

}
